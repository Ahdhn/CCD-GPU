#include "hip/hip_runtime.h"
#include <ccdgpu/helper.cuh>

#include <assert.h>
#include <ctype.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <iostream>

// #include <gputi/book.h>
// #include <gputi/io.h>
#include <ccdgpu/root_finder.cuh>
#include <ccdgpu/timer.hpp>
#include <gpubf/io.cuh>

#include <ccdgpu/record.hpp>
#include <gpubf/simulation.cuh>

using namespace std;
using namespace ccd;
using namespace ccdgpu;

__global__ void split_overlaps(const int2 *const overlaps,
                               const ccdgpu::Aabb *const boxes, int N,
                               int2 *vf_overlaps, int2 *ee_overlaps,
                               int *vf_count, int *ee_count) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= N)
    return;

  int minner = min(overlaps[tid].x, overlaps[tid].y);
  int maxxer = max(overlaps[tid].x, overlaps[tid].y);
  int3 avids = boxes[minner].vertexIds;
  int3 bvids = boxes[maxxer].vertexIds;

  if (is_vertex(avids) && is_face(bvids)) {
    int i = atomicAdd(vf_count, 1);
    vf_overlaps[i].x = minner;
    vf_overlaps[i].y = maxxer;

  } else if (is_edge(avids) && is_edge(bvids)) {
    int j = atomicAdd(ee_count, 1);
    ee_overlaps[j].x = minner;
    ee_overlaps[j].y = maxxer;
  }
}

__global__ void addData(const int2 *const overlaps,
                        const ccdgpu::Aabb *const boxes,
                        const ccd::Scalar *const V0,
                        const ccd::Scalar *const V1, int Vrows, int N,
                        ccd::Scalar ms, ccd::CCDdata *data) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= N)
    return;

  data[tid].ms = ms;

  // printf("vf_count %i, ee_count %i", *vf_count, *ee_count);

  int minner = min(overlaps[tid].x, overlaps[tid].y);
  int maxxer = max(overlaps[tid].x, overlaps[tid].y);
  int3 avids = boxes[minner].vertexIds;
  int3 bvids = boxes[maxxer].vertexIds;

  // data[tid].v0s[0] = a[8 * tid + 0].x;
  // data[tid].v1s[0] = a[8 * tid + 1].x;
  // data[tid].v2s[0] = a[8 * tid + 2].x;
  // data[tid].v3s[0] = a[8 * tid + 3].x;
  // data[tid].v0e[0] = a[8 * tid + 4].x;
  // data[tid].v1e[0] = a[8 * tid + 5].x;
  // data[tid].v2e[0] = a[8 * tid + 6].x;
  // data[tid].v3e[0] = a[8 * tid + 7].x;

  // data[tid].v0s[1] = a[8 * tid + 0].y;
  // data[tid].v1s[1] = a[8 * tid + 1].y;
  // data[tid].v2s[1] = a[8 * tid + 2].y;
  // data[tid].v3s[1] = a[8 * tid + 3].y;
  // data[tid].v0e[1] = a[8 * tid + 4].y;
  // data[tid].v1e[1] = a[8 * tid + 5].y;
  // data[tid].v2e[1] = a[8 * tid + 6].y;
  // data[tid].v3e[1] = a[8 * tid + 7].y;

  // data[tid].v0s[2] = a[8 * tid + 0].z;
  // data[tid].v1s[2] = a[8 * tid + 1].z;
  // data[tid].v2s[2] = a[8 * tid + 2].z;
  // data[tid].v3s[2] = a[8 * tid + 3].z;
  // data[tid].v0e[2] = a[8 * tid + 4].z;
  // data[tid].v1e[2] = a[8 * tid + 5].z;
  // data[tid].v2e[2] = a[8 * tid + 6].z;
  // data[tid].v3e[2] = a[8 * tid + 7].z;

  if (is_vertex(avids) && is_face(bvids)) {
    // int i = atomicAdd(vf_count, 1);
    // queries[8 * tid + 0] = ccd::make_Scalar3(
    //     V0[avids.x + 0], V0[avids.x + Vrows], V0[avids.x + 2 * Vrows]);
    for (size_t i = 0; i < 3; i++) {
      data[tid].v0s[i] = V0[avids.x + i * Vrows];
      data[tid].v1s[i] = V0[bvids.x + i * Vrows];
      data[tid].v2s[i] = V0[bvids.y + i * Vrows];
      data[tid].v3s[i] = V0[bvids.z + i * Vrows];
      data[tid].v0e[i] = V1[avids.x + i * Vrows];
      data[tid].v1e[i] = V1[bvids.x + i * Vrows];
      data[tid].v2e[i] = V1[bvids.y + i * Vrows];
      data[tid].v3e[i] = V1[bvids.z + i * Vrows];
    }

    // queries[8 * tid + 1] = ccd::make_Scalar3(
    //     V0[bvids.x + 0], V0[bvids.x + Vrows], V0[bvids.x + 2 * Vrows]);
    // queries[8 * tid + 2] = ccd::make_Scalar3(
    //     V0[bvids.y + 0], V0[bvids.y + Vrows], V0[bvids.y + 2 * Vrows]);

    // queries[8 * tid + 3] = ccd::make_Scalar3(
    //     V0[bvids.z + 0], V0[bvids.z + Vrows], V0[bvids.z + 2 * Vrows]);
    // queries[8 * tid + 4] = ccd::make_Scalar3(
    //     V1[avids.x + 0], V1[avids.x + Vrows], V1[avids.x + 2 * Vrows]);
    // ;
    // queries[8 * tid + 5] = ccd::make_Scalar3(
    //     V1[bvids.x + 0], V1[bvids.x + Vrows], V1[bvids.x + 2 * Vrows]);
    // ;
    // queries[8 * tid + 6] = ccd::make_Scalar3(
    //     V1[bvids.y + 0], V1[bvids.y + Vrows], V1[bvids.y + 2 * Vrows]);
    // ;
    // queries[8 * tid + 7] = ccd::make_Scalar3(
    //     V1[bvids.z + 0], V1[bvids.z + Vrows], V1[bvids.z + 2 * Vrows]);
    // ;
  } else if (is_edge(avids) && is_edge(bvids)) {
    // int j = atomicAdd(ee_count, 1);

    for (size_t i = 0; i < 3; i++) {
      data[tid].v0s[i] = V0[avids.x + i * Vrows];
      data[tid].v1s[i] = V0[avids.y + i * Vrows];
      data[tid].v2s[i] = V0[bvids.x + i * Vrows];
      data[tid].v3s[i] = V0[bvids.y + i * Vrows];
      data[tid].v0e[i] = V1[avids.x + i * Vrows];
      data[tid].v1e[i] = V1[avids.y + i * Vrows];
      data[tid].v2e[i] = V1[bvids.x + i * Vrows];
      data[tid].v3e[i] = V1[bvids.y + i * Vrows];
    }

    // queries[8 * tid + 0] = ccd::make_Scalar3(
    //     V0[avids.x + 0], V0[avids.x + Vrows], V0[avids.x + 2 * Vrows]);
    // ;

    // queries[8 * tid + 1] = ccd::make_Scalar3(
    //     V0[avids.y + 0], V0[avids.y + Vrows], V0[avids.y + 2 * Vrows]);
    // ;

    // queries[8 * tid + 2] = ccd::make_Scalar3(
    //     V0[bvids.x + 0], V0[bvids.x + Vrows], V0[bvids.x + 2 * Vrows]);
    // ;

    // queries[8 * tid + 3] = ccd::make_Scalar3(
    //     V0[bvids.y + 0], V0[bvids.y + Vrows], V0[bvids.y + 2 * Vrows]);
    // ;

    // queries[8 * tid + 4] = ccd::make_Scalar3(
    //     V1[avids.x + 0], V1[avids.x + Vrows], V1[avids.x + 2 * Vrows]);
    // ;

    // queries[8 * tid + 5] = ccd::make_Scalar3(
    //     V1[avids.y + 0], V1[avids.y + Vrows], V1[avids.y + 2 * Vrows]);
    // ;

    //   queries[8 * tid + 6] = ccd::make_Scalar3(
    //       V1[bvids.x + 0], V1[bvids.x + Vrows], V1[bvids.x + 2 * Vrows]);
    //   ;

    //   queries[8 * tid + 7] = ccd::make_Scalar3(
    //       V1[bvids.y + 0], V1[bvids.y + Vrows], V1[bvids.y + 2 * Vrows]);
    //   ;
  } else
    assert(0);
}

bool is_file_exist(const char *fileName) {
  ifstream infile(fileName);
  return infile.good();
}

void run_narrowphase(int2 *d_overlaps, Aabb *d_boxes, int count,
                     ccd::Scalar *d_vertices_t0, ccd::Scalar *d_vertices_t1,
                     int Vrows, int threads, int max_iter, ccd::Scalar tol,
                     ccd::Scalar ms, bool use_ms, bool allow_zero_toi,
                     vector<int> &result_list, ccd::Scalar &toi, Record &r) {

  toi = 1.0;

  int *d_vf_count;
  int *d_ee_count;
  hipMalloc((void **)&d_vf_count, sizeof(int));
  hipMalloc((void **)&d_ee_count, sizeof(int));

  int2 *d_vf_overlaps;
  int2 *d_ee_overlaps;

  int start_id = 0;
  int size = count;

  // double tavg = 0;
  // double tmp_tall = 0;

  while (1) {

    int remain = size - start_id;
    if (remain <= 0 || toi == 0)
      break;
    printf("remain %i, start_id %i\n", remain, start_id);

    int tmp_nbr = std::min(remain, MAX_OVERLAP_SIZE);

    r.Start("splitOverlaps", /*gpu=*/true);
    hipMemset(d_vf_count, 0, sizeof(int));
    hipMemset(d_ee_count, 0, sizeof(int));
    gpuErrchk(hipGetLastError());

    hipMalloc((void **)&d_vf_overlaps, sizeof(int2) * tmp_nbr);
    hipMalloc((void **)&d_ee_overlaps, sizeof(int2) * tmp_nbr);
    gpuErrchk(hipGetLastError());

    split_overlaps<<<tmp_nbr / threads + 1, threads>>>(
        d_overlaps + start_id, d_boxes, tmp_nbr, d_vf_overlaps, d_ee_overlaps,
        d_vf_count, d_ee_count);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    r.Stop();

    r.Start("createDataList", /*gpu=*/true);
    int vf_size;
    int ee_size;
    hipMemcpy(&vf_size, d_vf_count, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&ee_size, d_ee_count, sizeof(int), hipMemcpyDeviceToHost);
    cout << "vf_size " << vf_size << " ee_size " << ee_size << endl;
    gpuErrchk(hipGetLastError());

    CCDdata *d_ee_data_list;
    CCDdata *d_vf_data_list;

    size_t ee_data_size = sizeof(CCDdata) * ee_size;
    size_t vf_data_size = sizeof(CCDdata) * vf_size;

    hipMalloc((void **)&d_ee_data_list, ee_data_size);
    hipMalloc((void **)&d_vf_data_list, vf_data_size);
    printf("ee_data_size %llu\n", ee_data_size);
    printf("vf_data_size %llu\n", vf_data_size);
    gpuErrchk(hipGetLastError());

    addData<<<ee_size / threads + 1, threads>>>(
        d_ee_overlaps, d_boxes, d_vertices_t0, d_vertices_t1, Vrows, ee_size,
        ms, d_ee_data_list);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    addData<<<vf_size / threads + 1, threads>>>(
        d_vf_overlaps, d_boxes, d_vertices_t0, d_vertices_t1, Vrows, vf_size,
        ms, d_vf_data_list);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    r.Stop();

    printf("vf_size %i, ee_size %i\n", vf_size, ee_size);

    // int size = count;
    // cout << "data loaded, size " << queries.size() <<
    // endl;
    cout << "data loaded, size " << size << endl;

    // result_list.resize(size);

    int parallel = 64;
    printf("run_memory_pool_ccd using %i threads\n", parallel);
    r.Start("run_memory_pool_ccd (narrowphase)",
            /*gpu=*/true);
    // toi = 1;
    run_memory_pool_ccd(d_vf_data_list, vf_size, /*is_edge_edge=*/false,
                        result_list, parallel, max_iter, tol, use_ms,
                        allow_zero_toi, toi);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    printf("toi after vf %e\n", toi);
    // printf("time after vf %.6f\n", tmp_tall);

    run_memory_pool_ccd(d_ee_data_list, ee_size, /*is_edge_edge=*/true,
                        result_list, parallel, max_iter, tol, use_ms,
                        allow_zero_toi, toi);
    gpuErrchk(hipGetLastError());
    printf("toi after ee %e\n", toi);
    // printf("time after ee %.6f\n", tmp_tall);
    r.Stop();

    gpuErrchk(hipFree(d_vf_overlaps));
    gpuErrchk(hipFree(d_ee_overlaps));

    start_id += tmp_nbr;
  }
  gpuErrchk(hipFree(d_vf_count));
  gpuErrchk(hipFree(d_ee_count));
}

void run_ccd(const vector<Aabb> boxes, const Eigen::MatrixXd &vertices_t0,
             const Eigen::MatrixXd &vertices_t1, ccdgpu::Record &r, int N,
             int &nbox, int &parallel, int &devcount,
             vector<pair<int, int>> &overlaps, vector<int> &result_list,
             bool &use_ms, bool &allow_zero_toi, ccd::Scalar &ms,
             ccd::Scalar &toi) {
  int2 *d_overlaps;
  int *d_count;
  int threads = 32; // HARDCODING THREADS FOR NOW
  r.Start("run_sweep_sharedqueue (broadphase)", /*gpu=*/true);
  run_sweep_sharedqueue(boxes.data(), N, nbox, overlaps, d_overlaps, d_count,
                        threads, devcount);
  r.Stop();
  threads = 1024;
  gpuErrchk(hipGetLastError());
  printf("Threads now %i\n", threads);

  r.Start("copyBoxesToGpu", /*gpu=*/true);
  // copy overlap count
  int count;
  gpuErrchk(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
  printf("Count %i\n", count);
  gpuErrchk(hipGetLastError());

  // Allocate boxes to GPU
  Aabb *d_boxes;
  hipMalloc((void **)&d_boxes, sizeof(Aabb) * N);
  hipMemcpy(d_boxes, boxes.data(), sizeof(Aabb) * N, hipMemcpyHostToDevice);
  gpuErrchk(hipGetLastError());
  r.Stop();

  r.Start("copyVerticesToGpu", /*gpu=*/true);
  printf("Copying vertices\n");
  ccd::Scalar *d_vertices_t0;
  ccd::Scalar *d_vertices_t1;
  hipMalloc((void **)&d_vertices_t0, sizeof(ccd::Scalar) * vertices_t0.size());
  hipMalloc((void **)&d_vertices_t1, sizeof(ccd::Scalar) * vertices_t1.size());
  hipMemcpy(d_vertices_t0, vertices_t0.data(),
             sizeof(ccd::Scalar) * vertices_t0.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_vertices_t1, vertices_t1.data(),
             sizeof(ccd::Scalar) * vertices_t1.size(), hipMemcpyHostToDevice);
  r.Stop();
  int Vrows = vertices_t0.rows();
  assert(Vrows == vertices_t1.rows());

  int max_iter = 1e6;
  ccd::Scalar tolerance = 1e-6;

  run_narrowphase(d_overlaps, d_boxes, count, d_vertices_t0, d_vertices_t1,
                  Vrows, threads, max_iter, tolerance, ms, use_ms,
                  allow_zero_toi, result_list, toi, r);

  gpuErrchk(hipGetLastError());

  gpuErrchk(hipFree(d_overlaps));
  gpuErrchk(hipFree(d_boxes));
  gpuErrchk(hipFree(d_vertices_t0));
  gpuErrchk(hipFree(d_vertices_t1));

  gpuErrchk(hipGetLastError());

  hipDeviceSynchronize();
}

void compute_toi_strategy(const Eigen::MatrixXd &V0, const Eigen::MatrixXd &V1,
                          const Eigen::MatrixXi &E, const Eigen::MatrixXi &F,
                          ccd::Scalar min_distance, int max_iter, int tolerance,
                          ccd::Scalar &earliest_toi) {

  vector<ccdgpu::Aabb> boxes;
  constructBoxes(V0, V1, E, F, boxes);
  int N = boxes.size();
  int nbox = 0;
  int devcount = 1;

  vector<pair<int, int>> overlaps;
  vector<int> result_list;

  // BROADPHASE
  int2 *d_overlaps;
  int *d_count;
  int threads = 32; // HARDCODING THREADS FOR NOW
  run_sweep_sharedqueue(boxes.data(), N, nbox, overlaps, d_overlaps, d_count,
                        threads, devcount);
  threads = 1024;
  gpuErrchk(hipGetLastError());
  printf("Threads now %i\n", threads);

  // copy overlap count
  int count;
  gpuErrchk(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
  printf("Count %i\n", count);
  gpuErrchk(hipGetLastError());

  // Allocate boxes to GPU
  Aabb *d_boxes;
  hipMalloc((void **)&d_boxes, sizeof(Aabb) * N);
  hipMemcpy(d_boxes, boxes.data(), sizeof(Aabb) * N, hipMemcpyHostToDevice);
  gpuErrchk(hipGetLastError());

  printf("Copying vertices\n");
  ccd::Scalar *d_vertices_t0;
  ccd::Scalar *d_vertices_t1;
  hipMalloc((void **)&d_vertices_t0, sizeof(ccd::Scalar) * V0.size());
  hipMalloc((void **)&d_vertices_t1, sizeof(ccd::Scalar) * V1.size());
  hipMemcpy(d_vertices_t0, V0.data(), sizeof(ccd::Scalar) * V0.size(),
             hipMemcpyHostToDevice);
  hipMemcpy(d_vertices_t1, V1.data(), sizeof(ccd::Scalar) * V1.size(),
             hipMemcpyHostToDevice);

  int Vrows = V0.rows();
  assert(Vrows == V1.rows());

  Record r;

  run_narrowphase(d_overlaps, d_boxes, count, d_vertices_t0, d_vertices_t1,
                  Vrows, threads, max_iter, /*tol=*/tolerance,
                  /*ms=*/min_distance,
                  /*use_ms=*/false,
                  /*allow_zero_toi=*/true, result_list, earliest_toi, r);

  if (earliest_toi < 1e-6) {
    run_narrowphase(d_overlaps, d_boxes, count, d_vertices_t0, d_vertices_t1,
                    Vrows, threads, max_iter, /*tol=*/tolerance,
                    /*ms=*/min_distance,
                    /*use_ms=*/false,
                    /*allow_zero_toi=*/true, result_list, earliest_toi, r);
  }
}