#include "hip/hip_runtime.h"
#include <ccdgpu/helper.cuh>

#include <assert.h>
#include <ctype.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
#include <iostream>

// #include <gputi/timer.cuh>
#include <gputi/book.h>
#include <gputi/io.h>
// #include <gputi/read_rational_csv.cuh>
#include <gputi/root_finder.cuh>
#include <gputi/timer.hpp>

#include <ccdgpu/record.hpp>
#include <gpubf/simulation.cuh>

using namespace std;
using namespace ccd;
using namespace ccdgpu;

__global__ void addData(int2 *overlaps, Aabb *boxes, ccd::Scalar *V0,
                        ccd::Scalar *V1, int Vrows, int N,
                        ccd::Scalar3 *queries) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= N)
    return;

  int minner = min(overlaps[tid].x, overlaps[tid].y);
  int maxxer = max(overlaps[tid].x, overlaps[tid].y);
  int3 avids = boxes[minner].vertexIds;
  int3 bvids = boxes[maxxer].vertexIds;

  if (is_vertex(avids) && is_face(bvids)) {
    // auto vertex_start = V0.cast<float>().row(avids.x);
    // auto vertex_start = ccd::make_Scalar3(V0[avids.x + 0], V0[avids.x +
    // Vrows],
    //                                       V0[avids.x + 2 * Vrows]);
    // // // Triangle at t = 0
    // auto face_vertex0_start = ccd::make_Scalar3(
    //     V0[bvids.x + 0], V0[bvids.x + Vrows], V0[bvids.x + 2 * Vrows]);
    // auto face_vertex1_start = ccd::make_Scalar3(
    //     V0[bvids.y + 0], V0[bvids.y + Vrows], V0[bvids.y + 2 * Vrows]);
    // auto face_vertex2_start = ccd::make_Scalar3(
    //     V0[bvids.z + 0], V0[bvids.z + Vrows], V0[bvids.z + 2 * Vrows]);
    // // // Point at t=1
    // auto vertex_end = ccd::make_Scalar3(V1[avids.x + 0], V1[avids.x + Vrows],
    //                                     V1[avids.x + 2 * Vrows]);
    // // // Triangle at t = 1
    // auto face_vertex0_end = ccd::make_Scalar3(
    //     V1[bvids.x + 0], V1[bvids.x + Vrows], V1[bvids.x + 2 * Vrows]);
    // auto face_vertex1_end = ccd::make_Scalar3(
    //     V1[bvids.y + 0], V1[bvids.y + Vrows], V1[bvids.y + 2 * Vrows]);
    // auto face_vertex2_end = ccd::make_Scalar3(
    //     V1[bvids.z + 0], V1[bvids.z + Vrows], V1[bvids.z + 2 * Vrows]);

    queries[8 * tid + 0] = ccd::make_Scalar3(
        V0[avids.x + 0], V0[avids.x + Vrows], V0[avids.x + 2 * Vrows]);

    queries[8 * tid + 1] = ccd::make_Scalar3(
        V0[bvids.x + 0], V0[bvids.x + Vrows], V0[bvids.x + 2 * Vrows]);
    queries[8 * tid + 2] = ccd::make_Scalar3(
        V0[bvids.y + 0], V0[bvids.y + Vrows], V0[bvids.y + 2 * Vrows]);

    queries[8 * tid + 3] = ccd::make_Scalar3(
        V0[bvids.z + 0], V0[bvids.z + Vrows], V0[bvids.z + 2 * Vrows]);
    queries[8 * tid + 4] = ccd::make_Scalar3(
        V1[avids.x + 0], V1[avids.x + Vrows], V1[avids.x + 2 * Vrows]);
    ;
    queries[8 * tid + 5] = ccd::make_Scalar3(
        V1[bvids.x + 0], V1[bvids.x + Vrows], V1[bvids.x + 2 * Vrows]);
    ;
    queries[8 * tid + 6] = ccd::make_Scalar3(
        V1[bvids.y + 0], V1[bvids.y + Vrows], V1[bvids.y + 2 * Vrows]);
    ;
    queries[8 * tid + 7] = ccd::make_Scalar3(
        V1[bvids.z + 0], V1[bvids.z + Vrows], V1[bvids.z + 2 * Vrows]);
    ;

    // } else
    //   return;
  } else if (is_edge(avids) && is_edge(bvids)) {
    //     // Edge 1 at t=0
    // auto edge0_vertex0_start = ccd::make_Scalar3(
    //     V0[avids.x + 0], V0[avids.x + Vrows], V0[avids.x + 2 * Vrows]);
    // auto edge0_vertex1_start = ccd::make_Scalar3(
    //     V0[avids.y + 0], V0[avids.y + Vrows], V0[avids.y + 2 * Vrows]);
    // // Edge 2 at t=0
    // auto edge1_vertex0_start = ccd::make_Scalar3(
    //     V0[bvids.x + 0], V0[bvids.x + Vrows], V0[bvids.x + 2 * Vrows]);
    // auto edge1_vertex1_start = ccd::make_Scalar3(
    //     V0[bvids.y + 0], V0[bvids.y + Vrows], V0[bvids.y + 2 * Vrows]);
    // // Edge 1 at t=1
    // auto edge0_vertex0_end = ccd::make_Scalar3(
    //     V1[avids.x + 0], V1[avids.x + Vrows], V1[avids.x + 2 * Vrows]);
    // auto edge0_vertex1_end = ccd::make_Scalar3(
    //     V1[avids.y + 0], V1[avids.y + Vrows], V1[avids.y + 2 * Vrows]);
    // // Edge 2 at t=1
    // auto edge1_vertex0_end = ccd::make_Scalar3(
    //     V1[bvids.x + 0], V1[bvids.x + Vrows], V1[bvids.x + 2 * Vrows]);
    // auto edge1_vertex1_end = ccd::make_Scalar3(
    //     V1[bvids.y + 0], V1[bvids.y + Vrows], V1[bvids.y + 2 * Vrows]);

    queries[8 * tid + 0] = ccd::make_Scalar3(
        V0[avids.x + 0], V0[avids.x + Vrows], V0[avids.x + 2 * Vrows]);
    ;

    queries[8 * tid + 1] = ccd::make_Scalar3(
        V0[avids.y + 0], V0[avids.y + Vrows], V0[avids.y + 2 * Vrows]);
    ;

    queries[8 * tid + 2] = ccd::make_Scalar3(
        V0[bvids.x + 0], V0[bvids.x + Vrows], V0[bvids.x + 2 * Vrows]);
    ;

    queries[8 * tid + 3] = ccd::make_Scalar3(
        V0[bvids.y + 0], V0[bvids.y + Vrows], V0[bvids.y + 2 * Vrows]);
    ;

    queries[8 * tid + 4] = ccd::make_Scalar3(
        V1[avids.x + 0], V1[avids.x + Vrows], V1[avids.x + 2 * Vrows]);
    ;

    queries[8 * tid + 5] = ccd::make_Scalar3(
        V1[avids.y + 0], V1[avids.y + Vrows], V1[avids.y + 2 * Vrows]);
    ;

    queries[8 * tid + 6] = ccd::make_Scalar3(
        V1[bvids.x + 0], V1[bvids.x + Vrows], V1[bvids.x + 2 * Vrows]);
    ;

    queries[8 * tid + 7] = ccd::make_Scalar3(
        V1[bvids.y + 0], V1[bvids.y + Vrows], V1[bvids.y + 2 * Vrows]);
    ;
  } else
    assert(0);
}

void addData(const Aabb &a, const Aabb &b, const Eigen::MatrixXd &V0,
             const Eigen::MatrixXd &V1,
             vector<array<array<ccd::Scalar, 3>, 8>> &queries) {
  // auto is_vertex = [&](Aabb x){return x.vertexIds.y < 0 ;};
  // auto is_edge = [&](Aabb x){return !is_vertex(x) && x.vertexIds.z < 0 ;};
  // auto is_face = [&](Aabb x){return !is_vertex(x) && !is_edge(x);};

  // auto is_face = [&](Aabb x){return x.vertexIds.z >= 0;};
  // auto is_edge = [&](Aabb x){return x.vertexIds.z < 0 && x.vertexIds.y >= 0
  // ;}; auto is_vertex = [&](Aabb x){return x.vertexIds.z < 0  &&
  // x.vertexIds.y < 0;};

  if (is_vertex(a) && is_face(b)) {
    auto avids = a.vertexIds;
    auto bvids = b.vertexIds;
    // Point at t=0s
    auto vertex_start = V0.cast<ccd::Scalar>().row(avids.x);
    // // Triangle at t = 0
    auto face_vertex0_start = V0.cast<ccd::Scalar>().row(bvids.x);
    auto face_vertex1_start = V0.cast<ccd::Scalar>().row(bvids.y);
    auto face_vertex2_start = V0.cast<ccd::Scalar>().row(bvids.z);
    // // Point at t=1
    auto vertex_end = V1.cast<ccd::Scalar>().row(avids.x);
    // // Triangle at t = 1
    auto face_vertex0_end = V1.cast<ccd::Scalar>().row(bvids.x);
    auto face_vertex1_end = V1.cast<ccd::Scalar>().row(bvids.y);
    auto face_vertex2_end = V1.cast<ccd::Scalar>().row(bvids.z);

    array<array<ccd::Scalar, 3>, 8> tmp;
    tmp[0] = Vec3Conv(vertex_start);
    tmp[1] = Vec3Conv(face_vertex0_start);
    tmp[2] = Vec3Conv(face_vertex1_start);
    tmp[3] = Vec3Conv(face_vertex2_start);
    tmp[4] = Vec3Conv(vertex_end);
    tmp[5] = Vec3Conv(face_vertex0_end);
    tmp[6] = Vec3Conv(face_vertex1_end);
    tmp[7] = Vec3Conv(face_vertex2_end);
    queries.emplace_back(tmp);
  } else if (is_face(a) && is_vertex(b))
    return addData(b, a, V0, V1, queries);
  else if (is_edge(a) && is_edge(b)) {
    auto avids = a.vertexIds;
    auto bvids = b.vertexIds;
    //     // Edge 1 at t=0
    auto edge0_vertex0_start = V0.cast<ccd::Scalar>().row(avids.x);
    auto edge0_vertex1_start = V0.cast<ccd::Scalar>().row(avids.y);
    // // Edge 2 at t=0
    auto edge1_vertex0_start = V0.cast<ccd::Scalar>().row(bvids.x);
    auto edge1_vertex1_start = V0.cast<ccd::Scalar>().row(bvids.y);
    // // Edge 1 at t=1
    auto edge0_vertex0_end = V1.cast<ccd::Scalar>().row(avids.x);
    auto edge0_vertex1_end = V1.cast<ccd::Scalar>().row(avids.y);
    // // Edge 2 at t=1
    auto edge1_vertex0_end = V1.cast<ccd::Scalar>().row(bvids.x);
    auto edge1_vertex1_end = V1.cast<ccd::Scalar>().row(bvids.y);

    // queries.emplace_back(Vec3Conv(edge0_vertex0_start));
    array<array<ccd::Scalar, 3>, 8> tmp;
    tmp[0] = Vec3Conv(edge0_vertex0_start);
    tmp[1] = Vec3Conv(edge0_vertex1_start);
    tmp[2] = Vec3Conv(edge1_vertex0_start);
    tmp[3] = Vec3Conv(edge1_vertex1_start);
    tmp[4] = Vec3Conv(edge0_vertex0_end);
    tmp[5] = Vec3Conv(edge0_vertex1_end);
    tmp[6] = Vec3Conv(edge1_vertex0_end);
    tmp[7] = Vec3Conv(edge1_vertex1_end);
    queries.emplace_back(tmp);
  } else
    abort();
}

bool is_file_exist(const char *fileName) {
  ifstream infile(fileName);
  return infile.good();
}

__global__ void array_to_ccd(ccd::Scalar3 *a, int tmp_nbr, CCDdata *data) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= tmp_nbr)
    return;

#ifndef NO_CHECK_MS
  data[tid].ms = MINIMUM_SEPARATION_BENCHMARK;
#endif

  data[tid].v0s[0] = a[8 * tid + 0].x;
  data[tid].v1s[0] = a[8 * tid + 1].x;
  data[tid].v2s[0] = a[8 * tid + 2].x;
  data[tid].v3s[0] = a[8 * tid + 3].x;
  data[tid].v0e[0] = a[8 * tid + 4].x;
  data[tid].v1e[0] = a[8 * tid + 5].x;
  data[tid].v2e[0] = a[8 * tid + 6].x;
  data[tid].v3e[0] = a[8 * tid + 7].x;

  data[tid].v0s[1] = a[8 * tid + 0].y;
  data[tid].v1s[1] = a[8 * tid + 1].y;
  data[tid].v2s[1] = a[8 * tid + 2].y;
  data[tid].v3s[1] = a[8 * tid + 3].y;
  data[tid].v0e[1] = a[8 * tid + 4].y;
  data[tid].v1e[1] = a[8 * tid + 5].y;
  data[tid].v2e[1] = a[8 * tid + 6].y;
  data[tid].v3e[1] = a[8 * tid + 7].y;

  data[tid].v0s[2] = a[8 * tid + 0].z;
  data[tid].v1s[2] = a[8 * tid + 1].z;
  data[tid].v2s[2] = a[8 * tid + 2].z;
  data[tid].v3s[2] = a[8 * tid + 3].z;
  data[tid].v0e[2] = a[8 * tid + 4].z;
  data[tid].v1e[2] = a[8 * tid + 5].z;
  data[tid].v2e[2] = a[8 * tid + 6].z;
  data[tid].v3e[2] = a[8 * tid + 7].z;
}

void run_memory_pool_ccd(ccd::Scalar3 *V, int tmp_nbr, bool is_edge,
                         std::vector<int> &result_list, int parallel_nbr,
                         double &run_time, ccd::Scalar &toi) {
  unsigned nbr = tmp_nbr;
  result_list.resize(nbr);
  // host
  // CCDdata *data_list = new CCDdata[nbr];
  CCDdata *data_list;
  hipMalloc((void **)&data_list, sizeof(CCDdata) * nbr);
  array_to_ccd<<<nbr / parallel_nbr + 1, parallel_nbr>>>(V, nbr, data_list);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());
  printf("Finished array_to_ccd\n");

  int *res = new int[nbr];
  // MP_unit *units = new MP_unit[UNIT_SIZE];
  CCDConfig *config = new CCDConfig[1];
  config[0].err_in[0] =
      -1; // the input error bound calculate from the AABB of the whole mesh
  config[0].co_domain_tolerance = 1e-6; // tolerance of the co-domain
  // config[0].max_t = 1;                  // the upper bound of the time
  // interval
  config[0].toi = 1; // the maximal nbr of iterations
  config[0].mp_end = nbr;
  config[0].mp_start = 0;
  config[0].mp_remaining = nbr;

  // device
  CCDdata *d_data_list;
  // int *d_res;
  MP_unit *d_units;
  CCDConfig *d_config;

  size_t data_size = sizeof(CCDdata) * nbr;
  // size_t result_size = sizeof(int) * nbr;
  size_t unit_size = sizeof(MP_unit) * UNIT_SIZE;
  // int dbg_size=sizeof(ccd::Scalar)*8;

  hipMalloc(&d_data_list, data_size);
  // hipMalloc(&d_res, result_size);
  hipMalloc(&d_units, unit_size);
  hipMalloc(&d_config, sizeof(CCDConfig));

  hipMemcpy(d_data_list, data_list, data_size, hipMemcpyDeviceToDevice);
  hipMemcpy(d_config, config, sizeof(CCDConfig), hipMemcpyHostToDevice);
  gpuErrchk(hipGetLastError());

  ccd::Timer timer;
  hipProfilerStart();
  timer.start();
  printf("nbr: %i, parallel_nbr %i\n", nbr, parallel_nbr);
  initialize_memory_pool<<<nbr / parallel_nbr + 1, parallel_nbr>>>(d_units,
                                                                   nbr);
  hipDeviceSynchronize();
  compute_vf_tolerance_memory_pool<<<nbr / parallel_nbr + 1, parallel_nbr>>>(
      d_data_list, d_config, nbr);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

  printf("UNIT_SIZE: %llu\n", UNIT_SIZE);
  printf("EACH_LAUNCH_SIZE: %llu\n", EACH_LAUNCH_SIZE);
  // hipMemcpy(&toi, &d_config[0].toi, sizeof(ccd::Scalar),
  //            hipMemcpyDeviceToHost);
  // printf("toi init %.6f\n", toi);

  int nbr_per_loop = nbr;
  int start;
  int end;
  //   int inc = 0;
  while (nbr_per_loop > 0) {
    vf_ccd_memory_pool<<<nbr_per_loop / parallel_nbr + 1, parallel_nbr>>>(
        d_units, nbr, d_data_list, d_config);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    shift_queue_pointers<<<1, 1>>>(d_config);
    hipDeviceSynchronize();
    hipMemcpy(&nbr_per_loop, &d_config[0].mp_remaining, sizeof(int),
               hipMemcpyDeviceToHost);
    // hipMemcpy(&start, &d_config[0].mp_start, sizeof(int),
    //            hipMemcpyDeviceToHost);
    // hipMemcpy(&end, &d_config[0].mp_end, sizeof(int),
    // hipMemcpyDeviceToHost); hipMemcpy(&toi, &d_config[0].toi,
    // sizeof(ccd::Scalar),
    //            hipMemcpyDeviceToHost);
    // std::cout << "toi " << toi << std::endl;
    // printf("toi %.4f\n", toi);
    // printf("Start %i, End %i, Queue size: %i\n", start, end, nbr_per_loop);
    gpuErrchk(hipGetLastError());
    // printf("Queue size: %i\n", nbr_per_loop);
  }
  double tt = timer.getElapsedTimeInMicroSec();
  run_time = tt / 1000.0f;
  hipProfilerStop();
  gpuErrchk(hipGetLastError());

  // hipMemcpy(res, d_res, result_size, hipMemcpyDeviceToHost);
  hipMemcpy(&toi, &d_config[0].toi, sizeof(ccd::Scalar),
             hipMemcpyDeviceToHost);
  // hipMemcpy(dbg, d_dbg, dbg_size, hipMemcpyDeviceToHost);

  hipFree(data_list);
  hipFree(d_data_list);
  // hipFree(d_res);
  hipFree(d_units);
  hipFree(d_config);
  // hipFree(d_dbg);

  // for (size_t i = 0; i < nbr; i++) {
  //   result_list[i] = res[i];
  // }

  // delete[] res;
  // delete[] data_list;
  // delete[] units;
  delete[] config;
  // delete[] dbg;
  hipError_t ct = hipGetLastError();
  printf("******************\n%s\n************\n", hipGetErrorString(ct));

  return;
}

void run_ccd(vector<Aabb> boxes, const Eigen::MatrixXd &vertices_t0,
             const Eigen::MatrixXd &vertices_t1, Record &r, int N, int &nbox,
             int &parallel, int &devcount, vector<pair<int, int>> &overlaps,
             vector<int> &result_list, ccd::Scalar &toi) {
  int2 *d_overlaps;
  int *d_count;
  int threads = 0;
  r.Start("run_sweep_sharedqueue (broadphase)", /*gpu=*/true);
  run_sweep_sharedqueue(boxes.data(), N, nbox, overlaps, d_overlaps, d_count,
                        threads, devcount);
  gpuErrchk(hipDeviceSynchronize());
  r.Stop();
  gpuErrchk(hipGetLastError());
  printf("Threads now %i\n", threads);

  r.Start("broadphase -> narrowphase", /*gpu=*/true);
  // copy overlap count
  int count;
  gpuErrchk(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
  printf("Count %i\n", count);
  gpuErrchk(hipGetLastError());

  // Allocate boxes to GPU
  Aabb *d_boxes;
  hipMalloc((void **)&d_boxes, sizeof(Aabb) * N);
  hipMemcpy(d_boxes, boxes.data(), sizeof(Aabb) * N, hipMemcpyHostToDevice);
  gpuErrchk(hipGetLastError());

  ccd::Scalar3 *d_queries;
  size_t queries_size = sizeof(ccd::Scalar3) * 8 * count;
  cout << "queries size: " << queries_size << endl;
  hipMalloc((void **)&d_queries, queries_size);
  gpuErrchk(hipGetLastError());

  printf("Copying vertices\n");
  ccd::Scalar *d_vertices_t0;
  ccd::Scalar *d_vertices_t1;
  hipMalloc((void **)&d_vertices_t0, sizeof(ccd::Scalar) * vertices_t0.size());
  hipMalloc((void **)&d_vertices_t1, sizeof(ccd::Scalar) * vertices_t1.size());
  hipMemcpy(d_vertices_t0, vertices_t0.data(),
             sizeof(ccd::Scalar) * vertices_t0.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_vertices_t1, vertices_t1.data(),
             sizeof(ccd::Scalar) * vertices_t1.size(), hipMemcpyHostToDevice);

  int Vrows = vertices_t0.rows();
  assert(Vrows == vertices_t1.rows());

  gpuErrchk(hipGetLastError());
  addData<<<count / threads + 1, threads>>>(d_overlaps, d_boxes, d_vertices_t0,
                                            d_vertices_t1, Vrows, count,
                                            d_queries);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

  r.Stop();

  hipFree(d_overlaps);
  hipFree(d_boxes);
  hipFree(d_vertices_t0);
  hipFree(d_vertices_t1);

  hipDeviceSynchronize();

  // int size = queries.size();
  int size = count;
  // cout << "data loaded, size " << queries.size() << endl;
  cout << "data loaded, size " << size << endl;
  double tavg = 0;
  int max_query_cp_size = EACH_LAUNCH_SIZE;
  int start_id = 0;

  // result_list.resize(size);
  double tmp_tall;
  bool is_edge_edge = true;

  printf("run_memory_pool_ccd using %i threads\n", parallel);
  r.Start("run_memory_pool_ccd (narrowphase)", /*gpu=*/true);
  run_memory_pool_ccd(d_queries, size, is_edge_edge, result_list, parallel,
                      tmp_tall, toi);

  r.Stop();

  tavg += tmp_tall;
  cout << "tot time " << tavg << endl;
  tavg /= size;
  cout << "avg time " << tavg << endl;

  cout << "toi " << toi << endl;
}