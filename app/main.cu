
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <igl/readOBJ.h>
#include <igl/readPLY.h>
#include <igl/edges.h>

#include <gpubf/simulation.h>
#include <gpubf/groundtruth.h>
#include <gpubf/util.cuh>
#include <gpubf/io.cuh>

#define Vec3Conv(v) {v[0], v[1], v[2]}

// #include <gpubf/klee.cuh>

using namespace std;

void addData(
    const Aabb &a, 
    const Aabb &b, 
    const Eigen::MatrixXd& V0,
    const Eigen::MatrixXd& V1,
    std::vector<std::array<std::array<float, 3>, 8>>& queries)
{
    auto is_vertex = [&](Aabb x){return x.vertexIds.y < 0 ;};
    auto is_edge = [&](Aabb x){return !is_vertex(x) && x.vertexIds.z < 0 ;};
    auto is_face = [&](Aabb x){return !is_vertex(x) && !is_edge(x);};

    if (is_vertex(a) && is_face(b))
    {
        auto avids = a.vertexIds;
        auto bvids = b.vertexIds;
            // Point at t=0
        auto vertex_start = V0.cast<float>().row(avids.x);
        // // Triangle at t = 0
        auto face_vertex0_start = V0.cast<float>().row(bvids.x);
        auto face_vertex1_start = V0.cast<float>().row(bvids.y);
        auto face_vertex2_start = V0.cast<float>().row(bvids.z);
        // // Point at t=1
        auto vertex_end = V1.row(avids.x);
        // // Triangle at t = 1
        auto face_vertex0_end = V1.cast<float>().row(bvids.x);
        auto face_vertex1_end = V1.cast<float>().row(bvids.y);
        auto face_vertex2_end = V1.cast<float>().row(bvids.z);

        std::array<std::array<float, 3>, 8> tmp;
        tmp[0] = Vec3Conv(vertex_start);

    }
    else if (is_face(a) && is_vertex(b))
        return addData(b, a, V0, V1, queries);
    else if (is_edge(a) && is_edge(b))
    {
        auto avids = a.vertexIds;
        auto bvids = b.vertexIds;
        //     // Edge 1 at t=0
        auto edge0_vertex0_start = V0.cast<float>().row(avids.x);
        auto edge0_vertex1_start = V0.cast<float>().row(avids.y);
        // // Edge 2 at t=0
        auto edge1_vertex0_start = V0.cast<float>().row(bvids.x);
        auto edge1_vertex1_start = V0.cast<float>().row(bvids.y);
        // // Edge 1 at t=1
        auto edge0_vertex0_end = V1.cast<float>().row(avids.x);
        auto edge0_vertex1_end = V1.cast<float>().row(avids.y);
        // // Edge 2 at t=1
        auto edge1_vertex0_end = V1.cast<float>().row(bvids.x);
        auto edge1_vertex1_end = V1.cast<float>().row(bvids.y);
        
        // queries.emplace_back(Vec3Conv(edge0_vertex0_start));
    }
}


int main( int argc, char **argv )
{
    vector<char*> compare;

    const char* filet0 = argv[1];
    const char* filet1 = argv[2];
    
    vector<Aabb> boxes;
    Eigen::MatrixXd vertices_t0;
    Eigen::MatrixXd vertices_t1;
    Eigen::MatrixXi faces; 
    Eigen::MatrixXi edges;

    parseMesh(filet0, filet1, vertices_t0, vertices_t1, faces, edges);
    constructBoxes(vertices_t0, vertices_t1, faces, edges, boxes);
    int N = boxes.size();
    int nbox = 0;
    
    int o;
    while ((o = getopt (argc, argv, "c:n:b:")) != -1)
    {
        switch (o)
        {
            case 'c':
                optind--;
                for( ;optind < argc && *argv[optind] != '-'; optind++)
                {
                    compare.push_back(argv[optind]);
                    // compare_mathematica(overlaps, argv[optind]); 
                }
                break;
            case 'n':
                N = atoi(optarg);
                break;
            case 'b':
                nbox = atoi(optarg);
                break;
        }
    }

    vector<unsigned long> overlaps;
    run_sweep(boxes.data(), N, nbox, overlaps);

    std::vector<std::array<std::array<float, 3>, 8>> queries;
    for (int i=0; i < overlaps.size() / 2; i++)
    {
        int aid = overlaps[2*i];
        int bid = overlaps[2*i+1];

        Aabb a = boxes[aid];
        Aabb b = boxes[bid];  

        addData(a, b, vertices_t0, vertices_t1, queries);
    
    }

    
    // for (auto i : compare)
    // {
    //     compare_mathematica(overlaps, i);
    // }
    // cout << endl;

    // Mesh --> Boxes --> Broadphase --> (Boxes[2] ->float/double[8]) --> Narrowphase
    // Go back to old code and make overlaps as pairs
    
    // std::array<std::array<Scalar, 3>, 8> V = substract_ccd(all_V, i);
    // bool expected_result = results[i * 8];
    // queries.push_back(V);
    // expect_list.push_back(expected_result);

    // https://github.com/dbelgrod/broad-phase-benchmark/blob/main/src/narrowphase/symbolic.cpp
    // each overlap has 4 vertices over t0, t1 -> 8
    // just get vids and check them against V0 and V1
    // the 3 is the x,y,z coord of the vertex

    // fill in the rest of gputi main() to finish integration
    // also fix that bug

    


}