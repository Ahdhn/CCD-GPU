
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <igl/readOBJ.h>
#include <igl/readPLY.h>
#include <igl/edges.h>

#include <gpubf/simulation.h>
#include <gpubf/groundtruth.h>
#include <gpubf/util.cuh>
#include <gpubf/io.cuh>


// #include <gpubf/klee.cuh>

using namespace std;


int main( int argc, char **argv )
{
    vector<char*> compare;

    const char* filet0 = argv[1];
    const char* filet1 = argv[2];
    
    vector<Aabb> boxes;
    parseMesh(filet0, filet1, boxes);
    int N = boxes.size();
    int nbox = 0;
    
    int o;
    while ((o = getopt (argc, argv, "c:n:b:")) != -1)
    {
        switch (o)
        {
            case 'c':
                optind--;
                for( ;optind < argc && *argv[optind] != '-'; optind++)
                {
                    compare.push_back(argv[optind]);
                    // compare_mathematica(overlaps, argv[optind]); 
                }
                break;
            case 'n':
                N = atoi(optarg);
                break;
            case 'b':
                nbox = atoi(optarg);
                break;
        }
    }

    vector<unsigned long> overlaps;
    run_sweep(boxes.data(), N, nbox, overlaps);
    for (auto i : compare)
    {
        compare_mathematica(overlaps, i);
    }
    cout << endl;

}