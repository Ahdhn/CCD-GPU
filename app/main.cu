
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <igl/readOBJ.h>
#include <igl/readPLY.h>
#include <igl/edges.h>

#include <gpubf/simulation.cuh>
#include <gpubf/groundtruth.h>
#include <gpubf/util.cuh>
#include <gpubf/io.cuh>

#define Vec3Conv(v) {v[0], v[1], v[2]}

// #include <gpubf/klee.cuh>

using namespace std;

void addData(
    const Aabb &a, 
    const Aabb &b, 
    const Eigen::MatrixXd& V0,
    const Eigen::MatrixXd& V1,
    std::vector<std::array<std::array<float, 3>, 8>>& queries)
{
    // auto is_vertex = [&](Aabb x){return x.vertexIds.y < 0 ;};
    // auto is_edge = [&](Aabb x){return !is_vertex(x) && x.vertexIds.z < 0 ;};
    // auto is_face = [&](Aabb x){return !is_vertex(x) && !is_edge(x);};

    auto is_face = [&](Aabb x){return x.vertexIds.z >= 0;};
    auto is_edge = [&](Aabb x){return x.vertexIds.z < 0 && x.vertexIds.y >= 0 ;};
    auto is_vertex = [&](Aabb x){return x.vertexIds.z < 0  && x.vertexIds.y < 0;};

    if (is_vertex(a) && is_face(b))
    {
        auto avids = a.vertexIds;
        auto bvids = b.vertexIds;
            // Point at t=0s
        auto vertex_start = V0.cast<float>().row(avids.x);
        // // Triangle at t = 0
        auto face_vertex0_start = V0.cast<float>().row(bvids.x);
        auto face_vertex1_start = V0.cast<float>().row(bvids.y);
        auto face_vertex2_start = V0.cast<float>().row(bvids.z);
        // // Point at t=1
        auto vertex_end = V1.cast<float>().row(avids.x);
        // // Triangle at t = 1
        auto face_vertex0_end = V1.cast<float>().row(bvids.x);
        auto face_vertex1_end = V1.cast<float>().row(bvids.y);
        auto face_vertex2_end = V1.cast<float>().row(bvids.z);

        std::array<std::array<float, 3>, 8> tmp;
        tmp[0] = Vec3Conv(vertex_start);
        tmp[1] = Vec3Conv(face_vertex0_start);
        tmp[2] = Vec3Conv(face_vertex1_start);
        tmp[3] = Vec3Conv(face_vertex2_start);
        tmp[4] = Vec3Conv(vertex_end);
        tmp[5] = Vec3Conv(face_vertex0_end);
        tmp[6] = Vec3Conv(face_vertex1_end);
        tmp[7] = Vec3Conv(face_vertex2_end);
        queries.emplace_back(tmp);
    }
    else if (is_face(a) && is_vertex(b))
        return addData(b, a, V0, V1, queries);
    else if (is_edge(a) && is_edge(b))
    {
        auto avids = a.vertexIds;
        auto bvids = b.vertexIds;
        //     // Edge 1 at t=0
        auto edge0_vertex0_start = V0.cast<float>().row(avids.x);
        auto edge0_vertex1_start = V0.cast<float>().row(avids.y);
        // // Edge 2 at t=0
        auto edge1_vertex0_start = V0.cast<float>().row(bvids.x);
        auto edge1_vertex1_start = V0.cast<float>().row(bvids.y);
        // // Edge 1 at t=1
        auto edge0_vertex0_end = V1.cast<float>().row(avids.x);
        auto edge0_vertex1_end = V1.cast<float>().row(avids.y);
        // // Edge 2 at t=1
        auto edge1_vertex0_end = V1.cast<float>().row(bvids.x);
        auto edge1_vertex1_end = V1.cast<float>().row(bvids.y);
        
        // queries.emplace_back(Vec3Conv(edge0_vertex0_start));
        std::array<std::array<float, 3>, 8> tmp;
        tmp[0] = Vec3Conv(edge0_vertex0_start);
        tmp[1] = Vec3Conv(edge0_vertex1_start);
        tmp[2] = Vec3Conv(edge1_vertex0_start);
        tmp[3] = Vec3Conv(edge1_vertex1_start);
        tmp[4] = Vec3Conv(edge0_vertex0_end);
        tmp[5] = Vec3Conv(edge0_vertex1_end);
        tmp[6] = Vec3Conv(edge1_vertex0_end);
        tmp[7] = Vec3Conv(edge1_vertex1_end);
        queries.emplace_back(tmp);
    }
    else abort();
}

bool is_file_exist(const char *fileName)
{
    ifstream infile(fileName);
    return infile.good();
}

int main( int argc, char **argv )
{
    vector<char*> compare;

    char* filet0;
    char* filet1;

    filet0 = argv[1];
    if (is_file_exist(argv[2])) //CCD
        filet1 = argv[2];
    else //static CD
        filet1 = argv[1];
    
    vector<Aabb> boxes;
    Eigen::MatrixXd vertices_t0;
    Eigen::MatrixXd vertices_t1;
    Eigen::MatrixXi faces; 
    Eigen::MatrixXi edges;

    parseMesh(filet0, filet1, vertices_t0, vertices_t1, faces, edges);
    constructBoxes(vertices_t0, vertices_t1, faces, edges, boxes);
    int N = boxes.size();
    int nbox = 0;
    int parallel = 0;
    int devcount = 1;

    // std::copy(from_vector.begin(), from_vector.end(), to_vector.begin());
    
    int o;
    while ((o = getopt (argc, argv, "c:n:b:")) != -1)
    {
        switch (o)
        {
            case 'c':
                optind--;
                for( ;optind < argc && *argv[optind] != '-'; optind++)
                {
                    compare.push_back(argv[optind]);
                    // compare_mathematica(overlaps, argv[optind]); 
                }
                break;
            case 'n':
                N = atoi(optarg);
                break;
            case 'b':
                nbox = atoi(optarg);
                break;
        }
    }

    vector<pair<int,int>> overlaps;
    run_sweep_pieces(boxes.data(), N, nbox, overlaps, parallel, devcount);

    std::vector<std::array<std::array<float, 3>, 8>> queries;
    for (int i=0; i < overlaps.size(); i++)
    {
        int aid = overlaps[i].first;
        int bid = overlaps[i].second;

        Aabb a = boxes[aid];
        Aabb b = boxes[bid];  

        addData(a, b, vertices_t0, vertices_t1, queries);
    }
    printf("size: %i\n", queries.size());

    
    // for (auto i : compare)
    // {
    //     compare_mathematica(overlaps, i);
    // }
    // cout << endl;

    // Mesh --> Boxes --> Broadphase --> (Boxes[2] ->float/double[8]) --> Narrowphase
    // Go back to old code and make overlaps as pairs
    
    // std::array<std::array<Scalar, 3>, 8> V = substract_ccd(all_V, i);
    // bool expected_result = results[i * 8];
    // queries.push_back(V);
    // expect_list.push_back(expected_result);

    // https://github.com/dbelgrod/broad-phase-benchmark/blob/main/src/narrowphase/symbolic.cpp
    // each overlap has 4 vertices over t0, t1 -> 8
    // just get vids and check them against V0 and V1
    // the 3 is the x,y,z coord of the vertex

    // fill in the rest of gputi main() to finish integration
    // also fix that bug

    


}