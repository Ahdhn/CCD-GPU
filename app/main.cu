#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <igl/readOBJ.h>
#include <igl/readPLY.h>
#include <igl/edges.h>

#include <gpubf/simulation.cuh>
#include <gpubf/groundtruth.h>
#include <gpubf/util.cuh>
#include <gpubf/io.cuh>
#include <gpubf/aabb.cuh>

#include <gputi/root_finder.h>
#include <gputi/book.h>
#include <gputi/io.h>
#include <gputi/timer.cuh>
#include <gputi/timer.hpp>

using namespace ccd;

#define Vec3Conv(v) {v[0], v[1], v[2]}

// #include <gpubf/klee.cuh>

typedef float Scalar;

using namespace std;

void addData(
    const Aabb &a, 
    const Aabb &b, 
    const Eigen::MatrixXd& V0,
    const Eigen::MatrixXd& V1,
    vector<array<array<float, 3>, 8>>& queries)
{
    // auto is_vertex = [&](Aabb x){return x.vertexIds.y < 0 ;};
    // auto is_edge = [&](Aabb x){return !is_vertex(x) && x.vertexIds.z < 0 ;};
    // auto is_face = [&](Aabb x){return !is_vertex(x) && !is_edge(x);};

    // auto is_face = [&](Aabb x){return x.vertexIds.z >= 0;};
    // auto is_edge = [&](Aabb x){return x.vertexIds.z < 0 && x.vertexIds.y >= 0 ;};
    // auto is_vertex = [&](Aabb x){return x.vertexIds.z < 0  && x.vertexIds.y < 0;};

    if (is_vertex(a) && is_face(b))
    {
        auto avids = a.vertexIds;
        auto bvids = b.vertexIds;
            // Point at t=0s
        auto vertex_start = V0.cast<float>().row(avids.x);
        // // Triangle at t = 0
        auto face_vertex0_start = V0.cast<float>().row(bvids.x);
        auto face_vertex1_start = V0.cast<float>().row(bvids.y);
        auto face_vertex2_start = V0.cast<float>().row(bvids.z);
        // // Point at t=1
        auto vertex_end = V1.cast<float>().row(avids.x);
        // // Triangle at t = 1
        auto face_vertex0_end = V1.cast<float>().row(bvids.x);
        auto face_vertex1_end = V1.cast<float>().row(bvids.y);
        auto face_vertex2_end = V1.cast<float>().row(bvids.z);

        array<array<float, 3>, 8> tmp;
        tmp[0] = Vec3Conv(vertex_start);
        tmp[1] = Vec3Conv(face_vertex0_start);
        tmp[2] = Vec3Conv(face_vertex1_start);
        tmp[3] = Vec3Conv(face_vertex2_start);
        tmp[4] = Vec3Conv(vertex_end);
        tmp[5] = Vec3Conv(face_vertex0_end);
        tmp[6] = Vec3Conv(face_vertex1_end);
        tmp[7] = Vec3Conv(face_vertex2_end);
        queries.emplace_back(tmp);
    }
    else if (is_face(a) && is_vertex(b))
        return addData(b, a, V0, V1, queries);
    else if (is_edge(a) && is_edge(b))
    {
        auto avids = a.vertexIds;
        auto bvids = b.vertexIds;
        //     // Edge 1 at t=0
        auto edge0_vertex0_start = V0.cast<float>().row(avids.x);
        auto edge0_vertex1_start = V0.cast<float>().row(avids.y);
        // // Edge 2 at t=0
        auto edge1_vertex0_start = V0.cast<float>().row(bvids.x);
        auto edge1_vertex1_start = V0.cast<float>().row(bvids.y);
        // // Edge 1 at t=1
        auto edge0_vertex0_end = V1.cast<float>().row(avids.x);
        auto edge0_vertex1_end = V1.cast<float>().row(avids.y);
        // // Edge 2 at t=1
        auto edge1_vertex0_end = V1.cast<float>().row(bvids.x);
        auto edge1_vertex1_end = V1.cast<float>().row(bvids.y);
        
        // queries.emplace_back(Vec3Conv(edge0_vertex0_start));
        array<array<float, 3>, 8> tmp;
        tmp[0] = Vec3Conv(edge0_vertex0_start);
        tmp[1] = Vec3Conv(edge0_vertex1_start);
        tmp[2] = Vec3Conv(edge1_vertex0_start);
        tmp[3] = Vec3Conv(edge1_vertex1_start);
        tmp[4] = Vec3Conv(edge0_vertex0_end);
        tmp[5] = Vec3Conv(edge0_vertex1_end);
        tmp[6] = Vec3Conv(edge1_vertex0_end);
        tmp[7] = Vec3Conv(edge1_vertex1_end);
        queries.emplace_back(tmp);
    }
    else abort();
}

bool is_file_exist(const char *fileName)
{
    ifstream infile(fileName);
    return infile.good();
}



__global__ void run_parallel_vf_ccd_all(CCDdata *data,CCDConfig *config_in, bool *res, int size, Scalar *tois
)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= size) return;
    // copy the input queries to __device__
    CCDdata data_in;
    for (int i = 0; i < 3; i++)
    {
        data_in.v0s[i] = data[tx].v0s[i];
        data_in.v1s[i] = data[tx].v1s[i];
        data_in.v2s[i] = data[tx].v2s[i];
        data_in.v3s[i] = data[tx].v3s[i];
        data_in.v0e[i] = data[tx].v0e[i];
        data_in.v1e[i] = data[tx].v1e[i];
        data_in.v2e[i] = data[tx].v2e[i];
        data_in.v3e[i] = data[tx].v3e[i];
    }
    // copy the configurations to the shared memory
    __shared__ CCDConfig config;
    config.err_in[0]=config_in->err_in[0];
    config.err_in[1]=config_in->err_in[1];
    config.err_in[2]=config_in->err_in[2];
    config.co_domain_tolerance=config_in->co_domain_tolerance; // tolerance of the co-domain
    config.max_t=config_in->max_t; // the upper bound of the time interval
    config.max_itr=config_in->max_itr;// the maximal nbr of iterations
    CCDOut out;
    vertexFaceCCD(data_in,config, out);
    res[tx] = out.result;
    tois[tx] = 0;
}
__global__ void run_parallel_ee_ccd_all(CCDdata *data,CCDConfig *config_in, bool *res, int size, Scalar *tois
)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= size) return;
    // copy the input queries to __device__
    CCDdata data_in;
    for (int i = 0; i < 3; i++)
    {
        data_in.v0s[i] = data[tx].v0s[i];
        data_in.v1s[i] = data[tx].v1s[i];
        data_in.v2s[i] = data[tx].v2s[i];
        data_in.v3s[i] = data[tx].v3s[i];
        data_in.v0e[i] = data[tx].v0e[i];
        data_in.v1e[i] = data[tx].v1e[i];
        data_in.v2e[i] = data[tx].v2e[i];
        data_in.v3e[i] = data[tx].v3e[i];
    }
    // copy the configurations to the shared memory
    __shared__ CCDConfig config;
    config.err_in[0]=config_in->err_in[0];
    config.err_in[1]=config_in->err_in[1];
    config.err_in[2]=config_in->err_in[2];
    config.co_domain_tolerance=config_in->co_domain_tolerance; // tolerance of the co-domain
    config.max_t=config_in->max_t; // the upper bound of the time interval
    config.max_itr=config_in->max_itr;// the maximal nbr of iterations
    CCDOut out;
    edgeEdgeCCD(data_in,config, out);
    res[tx] = out.result;
    tois[tx] = 0;
}

__global__ void run_parallel_ms_vf_ccd_all(CCDdata *data,CCDConfig *config_in, bool *res, int size, Scalar *tois
)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= size) return;
    // copy the input queries to __device__
    CCDdata data_in;
    for (int i = 0; i < 3; i++)
    {
        data_in.v0s[i] = data[tx].v0s[i];
        data_in.v1s[i] = data[tx].v1s[i];
        data_in.v2s[i] = data[tx].v2s[i];
        data_in.v3s[i] = data[tx].v3s[i];
        data_in.v0e[i] = data[tx].v0e[i];
        data_in.v1e[i] = data[tx].v1e[i];
        data_in.v2e[i] = data[tx].v2e[i];
        data_in.v3e[i] = data[tx].v3e[i];
    }
    data_in.ms=data[tx].ms;
    // copy the configurations to the shared memory
    __shared__ CCDConfig config;
    config.err_in[0]=config_in->err_in[0];
    config.err_in[1]=config_in->err_in[1];
    config.err_in[2]=config_in->err_in[2];
    config.co_domain_tolerance=config_in->co_domain_tolerance; // tolerance of the co-domain
    config.max_t=config_in->max_t; // the upper bound of the time interval
    config.max_itr=config_in->max_itr;// the maximal nbr of iterations
    CCDOut out;
# ifdef NO_CHECK_MS
    vertexFaceCCD(data_in,config, out);
# else
    vertexFaceMinimumSeparationCCD(data_in,config, out);
#endif
    res[tx] = out.result;
    tois[tx] = 0;
}
__global__ void run_parallel_ms_ee_ccd_all(CCDdata *data,CCDConfig *config_in, bool *res, int size, Scalar *tois
)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= size) return;
    // copy the input queries to __device__
    CCDdata data_in;
    for (int i = 0; i < 3; i++)
    {
        data_in.v0s[i] = data[tx].v0s[i];
        data_in.v1s[i] = data[tx].v1s[i];
        data_in.v2s[i] = data[tx].v2s[i];
        data_in.v3s[i] = data[tx].v3s[i];
        data_in.v0e[i] = data[tx].v0e[i];
        data_in.v1e[i] = data[tx].v1e[i];
        data_in.v2e[i] = data[tx].v2e[i];
        data_in.v3e[i] = data[tx].v3e[i];
    }
    data_in.ms=data[tx].ms;
    // copy the configurations to the shared memory
    __shared__ CCDConfig config;
    config.err_in[0]=config_in->err_in[0];
    config.err_in[1]=config_in->err_in[1];
    config.err_in[2]=config_in->err_in[2];
    config.co_domain_tolerance=config_in->co_domain_tolerance; // tolerance of the co-domain
    config.max_t=config_in->max_t; // the upper bound of the time interval
    config.max_itr=config_in->max_itr;// the maximal nbr of iterations
    CCDOut out;
# ifdef NO_CHECK_MS
    edgeEdgeCCD(data_in,config, out);
# else
   edgeEdgeMinimumSeparationCCD(data_in,config, out);
#endif
    res[tx] = out.result;
    tois[tx] = 0;
}

void all_ccd_run(const std::vector<std::array<std::array<Scalar, 3>, 8>> &V, bool is_edge,
    std::vector<bool> &result_list, double &run_time, std::vector<Scalar> &time_impact, int parallel_nbr)
{
int nbr = V.size();
result_list.resize(nbr);
// host
CCDdata *data_list = new CCDdata[nbr];
for (int i = 0; i < nbr; i++)
{
data_list[i] = array_to_ccd( V[i]);
#ifndef NO_CHECK_MS
data_list[i].ms=MINIMUM_SEPARATION_BENCHMARK;
#endif
}

bool *res = new bool[nbr];
Scalar *tois = new Scalar[nbr];
CCDConfig *config=new CCDConfig[1];
config[0].err_in[0]=-1;// the input error bound calculate from the AABB of the whole mesh
config[0].co_domain_tolerance=1e-6; // tolerance of the co-domain
config[0].max_t=1; // the upper bound of the time interval
config[0].max_itr=1e6;// the maximal nbr of iterations

// device
CCDdata *d_data_list;
bool *d_res;
Scalar *d_tois;
CCDConfig *d_config;

int data_size = sizeof(CCDdata) * nbr;
int result_size = sizeof(bool) * nbr;
int time_size = sizeof(Scalar) * nbr;
// int dbg_size=sizeof(Scalar)*8;

hipMalloc(&d_data_list, data_size);
hipMalloc(&d_res, result_size);
hipMalloc(&d_tois, time_size);
hipMalloc(&d_config, sizeof(CCDConfig));

hipMemcpy(d_data_list, data_list, data_size, hipMemcpyHostToDevice);
hipMemcpy(d_config, config, sizeof(CCDConfig), hipMemcpyHostToDevice);

ccd::Timer timer;
hipProfilerStart();
timer.start();
#ifdef NO_CHECK_MS
if(is_edge){
run_parallel_ee_ccd_all<<<nbr / parallel_nbr + 1, parallel_nbr>>>( 
d_data_list,d_config, d_res, nbr, d_tois);
}
else{
run_parallel_vf_ccd_all<<<nbr / parallel_nbr + 1, parallel_nbr>>>( 
d_data_list,d_config, d_res, nbr, d_tois);
}
#else
if(is_edge){
run_parallel_ms_ee_ccd_all<<<nbr / parallel_nbr + 1, parallel_nbr>>>( 
d_data_list,d_config, d_res, nbr, d_tois);
}
else{
run_parallel_ms_vf_ccd_all<<<nbr / parallel_nbr + 1, parallel_nbr>>>( 
d_data_list,d_config, d_res, nbr, d_tois);
}
#endif

hipDeviceSynchronize();
double tt = timer.getElapsedTimeInMicroSec();
run_time = tt;
hipProfilerStop();

hipMemcpy(res, d_res, result_size, hipMemcpyDeviceToHost);
hipMemcpy(tois, d_tois, time_size, hipMemcpyDeviceToHost);
//hipMemcpy(dbg, d_dbg, dbg_size, hipMemcpyDeviceToHost);

hipFree(d_data_list);
hipFree(d_res);
hipFree(d_tois);
hipFree(d_config);
//hipFree(d_dbg);

for (int i = 0; i < nbr; i++)
{
result_list[i] = res[i];
}

time_impact.resize(nbr);

for (int i = 0; i < nbr; i++)
{
time_impact[i] = tois[i];
}
// std::cout << "dbg info\n"
//           << dbg[0] << "," << dbg[1] << "," << dbg[2] << "," << dbg[3] << "," << dbg[4] << "," << dbg[5] << "," << dbg[6] << "," << dbg[7] << std::endl;
delete[] res;
delete[] data_list;
delete[] tois;
delete[] config;
//delete[] dbg;
hipError_t ct = hipGetLastError();
printf("******************\n%s\n************\n", hipGetErrorString(ct));

return;
}


int main( int argc, char **argv )
{
    vector<char*> compare;

    char* filet0;
    char* filet1;

    filet0 = argv[1];
    if (is_file_exist(argv[2])) //CCD
        filet1 = argv[2];
    else //static CD
        filet1 = argv[1];
    
    vector<Aabb> boxes;
    Eigen::MatrixXd vertices_t0;
    Eigen::MatrixXd vertices_t1;
    Eigen::MatrixXi faces; 
    Eigen::MatrixXi edges;

    parseMesh(filet0, filet1, vertices_t0, vertices_t1, faces, edges);
    constructBoxes(vertices_t0, vertices_t1, faces, edges, boxes);
    int N = boxes.size();
    int nbox = 0;
    int parallel = 0;
    int devcount = 1;

    // std::copy(from_vector.begin(), from_vector.end(), to_vector.begin());
    
    int o;
    while ((o = getopt (argc, argv, "c:n:b:")) != -1)
    {
        switch (o)
        {
            case 'c':
                optind--;
                for( ;optind < argc && *argv[optind] != '-'; optind++)
                {
                    compare.push_back(argv[optind]);
                    // compare_mathematica(overlaps, argv[optind]); 
                }
                break;
            case 'n':
                N = atoi(optarg);
                break;
            case 'b':
                nbox = atoi(optarg);
                break;
        }
    }

    vector<pair<int,int>> overlaps;
    run_sweep_pieces(boxes.data(), N, nbox, overlaps, parallel, devcount);

    vector<array<array<float, 3>, 8>> queries;
    for (int i=0; i < overlaps.size(); i++)
    {
        int aid = overlaps[i].first;
        int bid = overlaps[i].second;

        Aabb a = boxes[aid];
        Aabb b = boxes[bid];  

        addData(a, b, vertices_t0, vertices_t1, queries);
    }
    int size = queries.size();
    cout << "data loaded, size " << queries.size() << endl;
    double tavg = 0;
    int max_query_cp_size = 1e7;
    int start_id = 0;

    
    vector<float> tois;
    vector<bool> result_list;
    result_list.resize(size);
    tois.resize(size);

    while (1)
    {
        vector<bool> tmp_results;
        vector<array<array<Scalar, 3>, 8>> tmp_queries;
        vector<Scalar> tmp_tois;

        int remain = size - start_id;
        double tmp_tall;

        if (remain <= 0)
            break;

        int tmp_nbr = min(remain, max_query_cp_size);
        tmp_results.resize(tmp_nbr);
        tmp_queries.resize(tmp_nbr);
        tmp_tois.resize(tmp_nbr);
        for (int i = 0; i < tmp_nbr; i++)
        {
            tmp_queries[i] = queries[start_id + i];
        }
        bool is_edge_edge = true;
        all_ccd_run(tmp_queries, is_edge_edge, tmp_results, tmp_tall, tmp_tois, parallel);

        tavg += tmp_tall;
        for (int i = 0; i < tmp_nbr; i++)
        {
            result_list[start_id + i] = tmp_results[i];
            tois[start_id + i] = tmp_tois[i];
        }

        start_id += tmp_nbr;
    }
    tavg /= size;
    cout << "avg time " << tavg << endl;
    
    for (auto i : compare)
    {
        compare_mathematica(overlaps, result_list, i);
    }
    cout << endl;

    // Mesh --> Boxes --> Broadphase --> (Boxes[2] ->float/double[8]) --> Narrowphase
    // Go back to old code and make overlaps as pairs
    
}